#include "hip/hip_runtime.h"
#include "kernel/head/analyse/hessienne.cuh"

static __global__ void add_cudarray(float * arr, uint pos, float value) {
	arr[pos] += value;
};

static void changer_weight(Train_t * train, uint set, uint wpos, float value) {
	add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, set*train->mdl->weights + wpos, value);
	printf("");
};

void calculer_tableau_hessienne(Hessienne_t * hessienne) {
	Opti_t * opti = hessienne->opti;
	Train_t * train = opti->train;
	Mdl_t * mdl = train->mdl;

	uint wsize = mdl->weights;
	uint sets = train->sets;

	train_set_input(train);

	//float dfxy[sets], dfx[sets];
	float f[sets], fx[sets], fy[sets], fxy[sets];

	//tableau : x=w1 y=w0
	//	d (d/dw0) / dw1

	Progressbar0(20)

	uint y, x;

	for (uint w0=0; w0 < wsize; w0++) {
		//df(x)
		train_forward(train, 0);
		score_dloss(train);
		train_backward(train, 0);
		
		
		//df(x+)
	}

	/*for (uint w0=0; w0 < wsize; w0++) {
		for (uint w1=0; w1 < wsize; w1++) {
			y = w0;
			x = w1;
			//Hess = [d/dwxdwy]

			
			//( df/dx(y+1e-5) - df/dx(y) ) / 1e-5
			//( f(x+,y+) - f(y+) - f(x+) + f() ) / 1e-10
		
			//( forward/backward(y+1e-5) - forward/backward(y) ) / 1e-5

			//forward/backward(y+1e-5)
			//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, set*wsize + w1, 1e-5);
			//CUDA_WAIT_KER()
			//train_forward(train, 0);
			//opti_dloss(hessienne->opti);
			//train_backward(train, 0);
			//SAFE_CUDA(hipMemcpy(&dfxy, train->_meand + set*wsize + w0, sizeof(float)*1, hipMemcpyDeviceToHost))

			//train_print_meands(train);

			//train_null_grad_meand(train);

			//forward/backward(y)
			//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, set*wsize + w1, - 1e-5);
			//CUDA_WAIT_KER()
			//train_forward(train, 0);
			//opti_dloss(hessienne->opti);
			//train_backward(train, 0);
			//SAFE_CUDA(hipMemcpy(&dfx, train->_meand + set*wsize + w0, sizeof(float)*1, hipMemcpyDeviceToHost))

			//hessienne->tableau[set*wsize*wsize + w0*wsize + w1] = (dfxy - dfx)/1e-5;

			//train_null_grad_meand(train);

			//f()
			train_forward(train, 0);
			opti_loss(opti);
			for (uint i=0; i < sets; i++) f[i] = opti->set_score[i];

			//f(x+)
			for (uint i=0; i < sets; i++) {
				//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, i*wsize + x, 1e-5);
				//CUDA_WAIT_KER()
				changer_weight(train, i, x, 1e-5);
			}
			train_forward(train, 0);
			opti_loss(opti);
			for (uint i=0; i < sets; i++) fx[i] = opti->set_score[i];

			//f(x+;y+)
			for (uint i=0; i < sets; i++) {
				//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, i*wsize + y, 1e-5);
				//CUDA_WAIT_KER()
				changer_weight(train, i, y, 1e-5);
			}
			train_forward(train, 0);
			opti_loss(opti);
			for (uint i=0; i < sets; i++) fxy[i] = opti->set_score[i];

			//f(y+)
			for (uint i=0; i < sets; i++) {
				//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, i*wsize + x, - 1e-5);
				//CUDA_WAIT_KER()
				changer_weight(train, i, x, - 1e-5);
			}
			train_forward(train, 0);
			opti_loss(opti);
			for (uint i=0; i < sets; i++) fy[i] = opti->set_score[i];

			//
			for (uint i=0; i < sets; i++) {
				//add_cudarray<<<dim3(1),dim3(1)>>>(train->_weight, i*wsize + y, - 1e-5);
				//CUDA_WAIT_KER()
				changer_weight(train, i, y, -1e-5);
			}

			//
			for (uint i=0; i < sets; i++) {
				//printf("%f %f %f %f\n", fxy[i], fy[i], fx[i], f[i]);
				hessienne->tableau[i*wsize*wsize + w0*wsize + w1] = (fxy[i] - fy[i] - fx[i] + f[i])/1e-10;
			}

			Progressbar((w0*wsize + w1)/(wsize*wsize), 20)
		}
	}*/

	Progressbar1(20)
}