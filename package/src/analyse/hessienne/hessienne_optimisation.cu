#include "hip/hip_runtime.h"
#include "kernel/head/analyse/hessienne.cuh"

//x -= H**(-1) * grad(x)

/*
	|x0|	| h0 h1 h2 |   | dx0 |
	|x1| -= | h3 h4 h5 | @ | dx1 |
	|x2|	| h6 h7 h8 |   | dx2 |
*/

static __global__ void appliquer_modification(float * weight, float * H_1, float * grad, uint sets, uint wsize, float alpha) {
	uint w = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (w < wsize) {
		float somme = 0;

		for (uint i=0; i < wsize; i++)
			somme += H_1[set*wsize*wsize + w*wsize + i] * grad[set*wsize + i];

		weight[set*wsize + w] -= alpha * somme;
	};
};

void opti_hessienne(Hessienne_t * hessienne) {
	Train_t * train = hessienne->opti->train;
	Mdl_t * mdl = train->mdl;

	uint wsize = mdl->weights;

	float alpha = 1.0;

	//	Obtention du gradient
	train_set_input(train);
	train_forward(train, 0);
	opti_dloss(hessienne->opti);
	train_backward(train, 0);

	//	
	appliquer_modification<<<dim3(KERN_DIV(wsize, 32),train->sets),dim3(32, 1)>>>(
		train->_weight, hessienne->inverse_par_set_d, train->_meand, train->sets, wsize, alpha
	);

	CUDA_WAIT_KER()
};