#include "kernel/head/analyse/hessienne.cuh"

/*
	M      1/M
| 1 2 3 | 1 0 0 |
| 4 5 6 | 0 1 0 |
| 7 8 9 | 0 0 1 |

Pivot de gauss

L1 = L1 - L0*4
...
Jusqu'a avoire Id a la place de M

*/

bool CPU_invert_hessienne(Hessienne_t * hessienne) {
	Train_t * train = hessienne->opti->train;
	Mdl_t * mdl = train->mdl;

	uint wsize = mdl->weights;

	uint n = wsize;

	float a[n*n];

	//
	for (uint set=0; set < train->sets; set++) {

		//	Id[n] pour l'inverse
		for (uint i=0; i < n; i++) {
			for (uint j=0; j < n; j++) {
				hessienne->inverse_par_set[set*wsize*wsize + i*n + j] = ( i == j ? 1 : 0);
				a[i*n + j] = hessienne->tableau[set*wsize*wsize + i*n + j];
			}
		}

		//On fait le pivot
		float coef;
		for (uint L=0; L < n; L++) {
			for (uint y=0; y < n; y++) {
				if (y == L) continue;

				if (a[L*n + L] == 0) {
					MSG("Non inversible");
					return false;
				}

				//Ly -= LL * coef
				coef = a[y*n + L] / a[L*n + L];
				for (uint k=0; k < n; k++) {
					a[y*n + k] -= a[L*n + k]*coef;
					hessienne->inverse_par_set[set*wsize*wsize + y*n + k] -= hessienne->inverse_par_set[set*wsize*wsize + L*n + k]*coef;
				}
			}
		};

		for (uint L=0; L < n; L++) {
			coef = a[L*n + L];
			if (coef == 0) {
				MSG("Non inversible");
				return false;
			}
			for (uint i=0; i < n; i++) {
				a[L*n + i] /= coef;
				hessienne->inverse_par_set[set*wsize*wsize + L*n + i] /= coef;
			}
		}
	}

	SAFE_CUDA(hipMemcpy(hessienne->inverse_par_set_d, hessienne->inverse_par_set, sizeof(float) * train->sets * wsize * wsize, hipMemcpyHostToDevice));

	return true;
};