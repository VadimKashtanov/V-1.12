#include "kernel/head/analyse/hessienne.cuh"

Hessienne_t * mk_hessienne(Opti_t * opti) {
	Hessienne_t * ret = (Hessienne_t*)malloc(sizeof(Hessienne_t));

	ret->opti = opti;

	uint sets = opti->train->sets;
	uint wsize = opti->train->mdl->weights;

	//ret->tableau_d = 0;
	ret->tableau = (float*)malloc(sizeof(float) * sets * wsize * wsize);

	ret->inverse_par_set_d = 0;
	ret->inverse_par_set = (float*)malloc(sizeof(float) * sets * wsize * wsize);

	return ret;
};

void cudmalloc_hessienne(Hessienne_t * hessienne) {
	uint sets = hessienne->opti->train->sets;
	uint wsize = hessienne->opti->train->mdl->weights;

	//SAFE_CUDA(hipMalloc((void**)&hessienne->tableau_d, sizeof(float) * sets * wszie * wszie))
	SAFE_CUDA(hipMalloc((void**)&hessienne->inverse_par_set_d, sizeof(float) * sets * wsize * wsize))
};

void free_hessienne(Hessienne_t * hessienne) {
	//if (hessienne->tableau_d) SAFE_CUDA(hipFree(hessienne->tableau_d));
	if (hessienne->inverse_par_set_d) SAFE_CUDA(hipFree(hessienne->inverse_par_set_d));

	free(hessienne->tableau);
	free(hessienne->inverse_par_set);

	free(hessienne);
};