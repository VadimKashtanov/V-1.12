#include "hip/hip_runtime.h"
#include "package/scores/meansquared/head/meansquared.cuh"

static __global__ void kernel_ms_ddloss(
	float * dd_grad, float * dd_var,
	uint total, uint ostart, uint lines, uint outs,
	uint sets)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		dd_var[pos] += dd_grad[pos];
	};
};

void meansquared_ddloss_th11(Train_t * train) {

	uint outpos = train->mdl->total - train->mdl->outputs;

	kernel_ms_ddloss<<<dim3(KERN_DIV(train->mdl->outputs, 32), KERN_DIV(train->data->lines, 32), train->sets),dim3(32, 32, 1)>>>(
		train->_dd_grad, train->_dd_var,
		train->mdl->total, outpos, train->data->lines, train->data->outputs,
		train->sets
	);
	
	CUDA_WAIT_KER();
};