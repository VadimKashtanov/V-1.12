#include "hip/hip_runtime.h"
#include "package/scores/meansquared/head/meansquared.cuh"

static __global__ void opti_kernel_ms_loss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs,
	uint sets)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		float g = var[pos];
		float w = output[line*outs + out];
		grad[pos] = pow(g - w, 2)/2;
	};
};

void meansquared_loss_th11(Train_t * train) {
	Mdl_t * mdl = train->mdl;

	uint outs = mdl->outputs;
	uint lines = train->data->lines;
	uint sets = train->sets;
	uint out_start = mdl->total - outs;

	//======================================================================

	//						compute score

	opti_kernel_ms_loss<<<dim3(KERN_DIV(outs, 32), KERN_DIV(lines, 32), sets),dim3(32,32,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, out_start, lines, outs,
		sets);
	CUDA_WAIT_KER();
};