#include "hip/hip_runtime.h"
#include "package/scores/meansquared/head/meansquared.cuh"

static __global__ void opti_kernel_sum_scores_over_lines(
	float * grad, float * var, float * output,
	float * score_one_line_d,
	uint total, uint lines, uint sets, uint ostart, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (out < outs)
	{
		float _sum_of_lines = 0;
		for (uint l=0; l < lines; l++) {
			_sum_of_lines += grad[l*sets*total + set*total + ostart + out];
		}
		score_one_line_d[set*outs + out] = _sum_of_lines;// / lines;
	};
};

static __global__ void opti_kernel_sum_scores_over_outputs(
	float * score_one_line_d, float * scores,
	uint total, uint sets, uint ostart, uint outs)
{
	uint set = blockIdx.x;

	uint start = set*outs + 0;
	float _sum_of_outs = 0;
	for (uint o=0; o < outs; o++) {
		_sum_of_outs += score_one_line_d[start];
		start++;
	}

	scores[set] = _sum_of_outs;// / outs;
};

void meansquared_score_th11(Train_t * train) {
	Mdl_t * mdl = train->mdl;

	uint outs = mdl->outputs;
	uint lines = train->data->lines;
	uint sets = train->sets;
	uint out_start = mdl->total - outs;

	//======================================================================
	//======================================================================

	//				sum over lines (only outputs)

	float * score_one_line_d;
	SAFE_CUDA(hipMalloc((void**)&score_one_line_d, sizeof(float) * sets * outs));	//all lines are sumed in one (only outputs)

	opti_kernel_sum_scores_over_lines<<<dim3(KERN_DIV(outs, 16), sets),dim3(16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		score_one_line_d,
		mdl->total, lines, sets, out_start, outs);
	CUDA_WAIT_KER();

	//======================================================================
	//======================================================================

	//		sum of output pixels

	opti_kernel_sum_scores_over_outputs<<<dim3(sets),dim3(1)>>>(
		score_one_line_d, train->set_score_d,
		mdl->total, sets, out_start, outs);
	CUDA_WAIT_KER();

	SAFE_CUDA(hipFree(score_one_line_d));

	//	Always copy set_score_d into Cpu Ram
	SAFE_CUDA(hipMemcpy(train->set_score, train->set_score_d, sizeof(float) * sets, hipMemcpyDeviceToHost));
};