#include "hip/hip_runtime.h"
#include "package/scores/meansquared/head/meansquared.cuh"

static __global__ void kernel_ms_dloss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs,
	uint sets)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		grad[pos] = var[pos] - output[line*outs + out];
	};
};

void meansquared_dloss_th11(Train_t * train) {
	uint outpos = train->mdl->total - train->mdl->outputs;

	kernel_ms_dloss<<<dim3(KERN_DIV(train->mdl->outputs, 32), KERN_DIV(train->data->lines, 32), train->sets),dim3(32, 32, 1)>>>(
		train->_grad, train->_var, train->data->output_d,
		train->mdl->total, outpos, train->data->lines, train->data->outputs,
		train->sets
	);
	CUDA_WAIT_KER();
};