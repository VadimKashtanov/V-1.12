#include "hip/hip_runtime.h"
#include "package/package.cuh"

//./mdl_weight_limits mdl.bin -3 3
//		ca va juste faire min(max(-3, weight), 3)

int main(int argc, char ** argv) {
	if (argc != 4)
		ERR("You have to give : mdl, borne0, borne1");

	FILE * fp = fopen(argv[1], "rb");
	Mdl_t * mdl = mdl_fp_load(fp);
	mdl_check_correctness(mdl);
	fclose(fp);

	float borne0 = atof(argv[2]), borne1 = atof(argv[3]);

	for (uint i=0; i < mdl->weights; i++) {
		if (mdl->weight[i] > borne1) mdl->weight[i] = borne1;
		if (mdl->weight[i] < borne0) mdl->weight[i] = borne0;
	};

	fp = fopen(argv[1], "wb");
	mdl_fp_write(mdl, fp);
	fclose(fp);
};