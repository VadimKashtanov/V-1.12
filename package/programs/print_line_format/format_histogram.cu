#include "hip/hip_runtime.h"
#include "package/programs/print_line_format/print_line_format.cuh"

static void max_and_min(float * arr, uint len, float * _max, float * _min) {
	float __max = 0;
	float __min = arr[0];

	for (uint i=0; i < len; i++) {
		if (arr[i] > __max)
			__max = arr[i];

		if (arr[i] < __min)
			__min = arr[i];
	}

	*_max = __max;
	*_min = __min;
};

void format_histogram(float * start_addres, uint size) {
	float _min, _max;

	max_and_min(start_addres, size, &_max, &_min);

	uint val;
	for (uint i=0; i < size; i++) {
		printf("%3.i | ", i);
		val = (uint)(round((start_addres[i] - _min)/(_max - _min) * 255.0));
		for (uint j=0; j < (uint)(round((start_addres[i] - _min)/(_max - _min) * 20.0)); j++)
			printf("\033[48;2;%i;%i;%im  \033[0m", val, val, val);
		for (uint j=(uint)(round((start_addres[i] - _min)/(_max - _min) * 20.0)); j < 20; j++)
			printf("  ");
		printf("  %.3g\n", start_addres[i]);
	}
};