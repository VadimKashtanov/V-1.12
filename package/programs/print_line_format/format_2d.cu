#include "hip/hip_runtime.h"
#include "package/programs/print_line_format/print_line_format.cuh"

static void max_and_min(float * arr, uint len, float * _max, float * _min) {
	float __max = 0;
	float __min = arr[0];

	for (uint i=0; i < len; i++) {
		if (arr[i] > __max)
			__max = arr[i];

		if (arr[i] < __min)
			__min = arr[i];
	}

	*_max = __max;
	*_min = __min;
};

void format_2dsquare(float * start_addres, uint size) {
	if (sqrt(size) != round(sqrt(size))) {
		ERR("Il faut un inputs qui a une racine carre entiere, afin de dessiner un carree")
	}

	uint len = (uint)sqrt(size);

	float _min, _max;

	max_and_min(start_addres, size, &_max, &_min);

	uint val;

	for (uint i=0; i < len; i++) {
		for (uint j=0; j < len; j++) {
			val = (uint)(round((start_addres[i*len + j] - _min)/(_max - _min) * 255.0));
			printf("\033[48;2;%i;%i;%im  \033[0m", val, val, val);
		}
		printf("\n");
	}
};