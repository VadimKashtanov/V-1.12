#include "hip/hip_runtime.h"
#include "package/package.cuh"

//	./optimize_mdl config_file

//	Regarder compile.py pour obtenire la maniere de configurer le fichier de config

/*

Algorithme:

Fix <model> <test_data>

for echope in echopes:
	for no_test in no_test_pass:

		batch = random() % batchs
		for same_batch in repeats:
			init()
			forward()
			dloss()
			backward()
			update()
			
	test()

	if max(score) < limit_score:
		break

	if echope % each_n_echopes == 0:
		select()
*/

char * mdl_file = 0;
char * data_file = 0;
//char * test_data_file = 0;
char * out_file = 0;

uint echopes = 1;
uint no_test_passs = 1;
uint same_start_seed_runs = 1;
uint repeats = 1;

uint sets = 1;
uint is_train_random = 1;

float limite_score = 0.0;

uint opti_algo = 0;
char ** opti_args_keys = 0;
char ** opti_args_values = 0;

uint score_algo = 0;
char ** score_args_keys = 0;
char ** score_args_values = 0;

uint echo_weights = 0;
uint echo_vars = 0;
uint echo_locds = 0;
uint echo_grads = 0;
uint echo_meands = 0;

uint read_uint(FILE * fp) {
	uint ret;
	fread(&ret, sizeof(uint), 1, fp);
	return ret;
};

float read_float(FILE * fp) {
	float ret;
	fread(&ret, sizeof(float), 1, fp);
	return ret;
};

char * read_str(FILE * fp) {
	uint len = read_uint(fp);

	char * ret = (char*)malloc(len + 1);
	fread(ret, sizeof(char), len, fp);
	ret[len] = '\0';

	return ret;
};

char ** read_strs(FILE * fp, uint len) {
	char ** ret = (char**)malloc(sizeof(char*) * len);

	uint _str_len;
	for (uint i=0; i < len; i++) {
		_str_len = read_uint(fp);
		ret[i] = (char*)malloc(len+1);
		fread(ret[i], sizeof(char), _str_len, fp);
		ret[i][_str_len] = '\0';
	};

	return ret;
};

void free_strs(char ** strs, uint len) {
	for (uint i=0; i < len; i++)
		free(strs[i]);
	free(strs);
};

int main(int argc, char ** argv) {
	assert(argc == 2);

	//
	//	**** Lancer le code python ****
	//

	char * command = (char*)malloc(100 + strlen(argv[1]));
	sprintf(command, "python3 -m package.programs.optimize_smart.main %s", argv[1]);
	if (system(command) != 0) ERR("Command failled");
	free(command);

	//Some variables to process code
	uint opti_args_len, score_args_len;

	//	File where config is written
	FILE * fp = fopen("package/programs/optimize_smart/tmpt", "rb");

	if (fp == 0)
		ERR("Le python ne c'est pas compilé comme il le faut")

	//
	//	**** Read all configuration parameters ****
	//

	mdl_file = read_str(fp);
	data_file = read_str(fp);
	//test_data_file = read_str(fp);
	out_file = read_str(fp);

	echopes = read_uint(fp);
	no_test_passs = read_uint(fp);
	same_start_seed_runs = read_uint(fp);
	repeats = read_uint(fp);

	sets = read_uint(fp);
	is_train_random = read_uint(fp);

	limite_score = read_float(fp);

	//opti
	opti_algo = read_uint(fp);
	opti_args_len = read_uint(fp);
	opti_args_keys = read_strs(fp, opti_args_len);
	opti_args_values = read_strs(fp, opti_args_len);

	//score
	score_algo = read_uint(fp);
	score_args_len = read_uint(fp);
	score_args_keys = read_strs(fp, score_args_len);
	score_args_values = read_strs(fp, score_args_len);

	echo_weights = read_uint(fp);
	echo_vars = read_uint(fp);
	echo_locds = read_uint(fp);
	echo_grads = read_uint(fp);
	echo_meands = read_uint(fp);

	//
	//	**** Build Train_t and all we need ****
	//
	//=====================================
	
		FILE * mdlfp = fopen(mdl_file, "rb");
		Mdl_t * mdl = mdl_fp_load(mdlfp);
		fclose(mdlfp);
	
	//=====================================
	
		//// Load to Ram and Vram
		Data_t * data = data_open(data_file);
		//Data_t * test_data = data_open(test_data_file);

		/*assert(data->lines == test_data->lines);
		assert(data->inputs == test_data->inputs);
		assert(data->outputs == test_data->outputs);*/

		FILE * data_fp = fopen(data_file, "rb");
		//FILE * test_data_fp = fopen(test_data_file, "rb");

		data_cudamalloc(data);
	
	//=====================================
	
		if (sets == 0)
			ERR("sets can't be = to 0")

		Train_t * train = mk_train(mdl, data, sets);

		if (is_train_random == 1) {
			train_random_weights(train, rand()%10000);
		} else {
			//	Random all not-0th sets
			if (sets > 1) train_random_weights_from_mdl(train, rand()%10000);

			//	The zero'th set have to be the mdl one
			train_inject_weight_from_mdl_to_one_set(train, 0);
		}
	
	//=====================================
	
		Opti_t * opti = opti_mk(train, score_algo, opti_algo);

		for (uint i=0; i < opti_args_len; i++)
			opti_opti_set_one_arg(opti, opti_args_keys[i], opti_args_values[i]);
		for (uint i=0; i < score_args_len; i++)
			opti_score_set_one_arg(opti, score_args_keys[i], score_args_values[i]);

	uint start_seed;
	uint batch_train;

	//uint best_set;
	//float best_score, old_best_score;

	//old_best_score = 100000.0;

	printf("# Starting trainning.\n");

/*#if __NVPROFIL__ == true
	hipProfilerStart();
#endif*/

	float score[train->sets];
	float _max;
	uint _max_id;

	for (uint lp=0; lp < echopes; lp++) {

		//
		//	Train with data
		//

		//data_cudamalloc(data);
		
		start_seed = rand() % 100000;

		for (uint i=0; i < train->sets; i++)
			score[i] = 0;

		for (uint same_start_seed=0; same_start_seed < same_start_seed_runs; same_start_seed++) {
			//Loop
			batch_train = rand() % data->batchs;

			//	Load a batch
			//StartTimer
			data_load_batch(data, data_fp, batch_train);
			data_cudamemcpy(data);
				
			//	Trainning Part
			for (uint i=0; i < repeats; i++) {
				//	Initialise correctly
				train_null_grad_meand(train);
				train_set_input(train);
			
				//	Forward and Backward
				train_forward(train, start_seed);
				opti_dloss(opti);
				train_backward(train, start_seed);

				//	Optimize
				opti_opti(opti);

				//	Compute Score on this
				//train_null_grad_output(train);
				opti_loss(opti);
				
				for (uint i=0; i < train->sets; i++) {
					//printf("%f\n", opti->set_score[i]);
					//raise(SIGINT);
					score[i] += opti->set_score[i];
					//opti->set_score[i] = 0;
				}

				//==== Prints ====
				if (echo_weights)
					train_print_weights(train);
				if (echo_vars)
					train_print_vars(train);
				if (echo_locds)
					train_print_locds(train);
				if (echo_grads)
					train_print_grads(train);
				if (echo_meands)
					train_print_meands(train);
			}
		}
		_max_id = 0;
		_max = score[0] / (same_start_seed_runs * repeats);
		for (uint i=0; i < train->sets; i++) {
			score[i] /= (same_start_seed_runs * repeats);
			if (score[i] < _max) {
				_max = score[i];
				_max_id = i;
			}
		}

		train_cpy_ws_to_mdl(train, _max_id);

		//data_free_cudamalloc(data);

		//
		//	Test Score with test_data (all batchs)
		//

		/*data_cudamalloc(test_data);
		train->data = test_data;

		find_best_set(test_data, test_data_fp, opti, train, &best_set, &best_score);
		if (best_score < old_best_score) {	//on cherche bien a minimiser Loss(f(x), w)
			old_best_score = best_score;

			//	On copy pour avoire tout le temps le meilleur et sauvgarder ceux qui ont les meilleurs scores
			train_cpy_ws_to_mdl(train, best_set);
		}

		data_free_cudamalloc(test_data);
		train->data = data;*/

		printf("Echope : %i/%i (best score:%f, set=%i)\n", lp+1, echopes, _max, _max_id);
	};

	mdlfp = fopen(out_file, "wb");
	mdl_fp_write(mdl, mdlfp);
	fclose(mdlfp);

	//
	//	**** Free all *****
	//

	//free malloc config params
	free(mdl_file);
	free(data_file);
	//free(test_data_file);
	free(out_file);

	free_strs(opti_args_keys, opti_args_len);
	free_strs(opti_args_values, opti_args_len);
	free_strs(score_args_keys, score_args_len);
	free_strs(score_args_values, score_args_len);

	if (system("rm package/programs/optimize_smart/tmpt") != 0) ERR("Command error");

	fclose(fp);

/*#if __NVPROFIL__ == true
	hipProfilerStop();
#endif*/

	data_free(data);
	train_free(train);
	mdl_free(mdl);
};