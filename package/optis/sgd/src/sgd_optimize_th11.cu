#include "hip/hip_runtime.h"
#include "package/optis/sgd/head/sgd.cuh"

static __global__
void sgd_kernel_th11(
	float sgd_alpha,
	uint weights, uint lines,
	float * weight, float * meand)
{
	uint w = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (w < weights)
		weight[set*weights + w] -= sgd_alpha * meand[set*weights + w] / lines;
};

void sgd_opti_th11(Train_t * train)
{
	Config_t * config = (Config_t*)train->opti;
	SGDData_t * ret = (SGDData_t*)config->ptr;

	ret->echopes++;
	float alpha = ret->alpha;

	if (alpha > 100000 || alpha < 0.0000001) ERR("alpha = %f", alpha);

	sgd_kernel_th11<<<dim3(KERN_DIV(train->mdl->weights, 16), train->sets),dim3(16,1)>>>(
		alpha,
		train->mdl->weights, train->data->lines,
		train->_weight, train->_meand
	);

	//opti_sgd_alpha *= ( 1 / ( 1 + 0.1/2000 * ret->echopes));

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//
//
//		Tester avec un probleme de beaucoup plus petite taille
//		Pas du Mnist, mais par exemple classifier 3 lettres A,B,C avec des images de taille 8x8
//		Pour kconvl 8x8->4x4->2x2->dot1d(4)->softmax(4)
//
//		1. Faire en python un petit programme qui ecrit les lettres en pixels avec l'IDLE
//		2. Faire un Data_t avec ces pixels
//		3. Cree le model simple et observer les trucs
//
//
//
//		Au lieux de fait un model qui predit plusieurs classes,
//		On train plusieur models d'une seule classe binaire.
//		Ex : On train le model qui predit un 0 ou un autre nombre que 0
//			puis On train le model qui predit un 1 ou un autre nombre que 1
//			puis le model qui predit un 2 ou un autre nombre que 2
//				..., jusqu'a 9
//		Apres on unie tout. On y ajoute un dot1->dot1d->softmax (en coupant le derniere dot1d->softmax des models precedants)
//		Bon ducoup ça train pas la diversité ensemble (des 0,1,2,3 .. n)
//		Mais ça va s'auto train juste apres tout seul.
//
//		(a la limite si les kernels bougent trop, on peut donner le resulats des model_precedant(input_qui_correspond) -> output)
//		(et le output sera mis dans le Data_t et puis plus tard, une fois train, on met ensemble tous les models)