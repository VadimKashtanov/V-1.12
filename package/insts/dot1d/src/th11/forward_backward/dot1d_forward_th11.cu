#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void dot1d_forward_th11(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint input_start, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (x < Yx) {

		uint Apos;// = time*sets*total + set*total + input_start;
		uint Bpos;// + _Yx*Ax;

		//uint __seed;
		//float value;

		float sum = 0;
		for (uint i=0; i < Ax; i++) {

			Apos = time*sets*total + set*total + input_start + i;
			Bpos = set*wsize + wstart + x + i*Yx;

			//__seed = Apos * (seed+1);

			//value = pseudo_randomf(__seed);

			//if ((value) >= drop_rate) {
			sum += var[Apos] * weight[Bpos];
			//}
		}
		sum += weight[set*wsize + wstart + Ax*Yx + x];
		
		float __locd;

		ACTIVATION_FORWARD(activ, sum, sum, __locd)

		var[time*sets*total + set*total + ystart + x] = sum;		//same assembler than putting it in if/else structure
		locd[time*sets*lsize + set*lsize + lstart + x] = __locd;
	}
};
