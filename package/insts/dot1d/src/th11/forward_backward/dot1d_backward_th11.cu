#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void dot1d_backward_th11(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (_Yx < Yx) {
		uint Apos = time*sets*total + set*total + istart;
		//uint weight_start = set*wsize + wstart;
		uint Bpos = set*wsize + wstart + _Yx;// _Yx*Ax;

		float dlds = locd[time*sets*lsize + set*lsize + lstart + _Yx] * grad[time*sets*total + set*total + ystart + _Yx];

		meand[set*wsize + wstart + Yx*Ax + _Yx] += dlds;

		for (uint i=0; i < Ax; i++) {
			//if (pseudo_randomf(Apos*(seed+1)) >= drop_rate) {
			atomicAdd(&grad[Apos], dlds * weight[Bpos]);
			atomicAdd(&meand[Bpos], dlds * var[Apos]);
			//}
			Apos++;
			Bpos += Yx;
		}
	}
};