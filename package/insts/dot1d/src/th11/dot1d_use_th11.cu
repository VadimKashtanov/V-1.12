#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void dot1d_use_th11(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x;

	if (_Yx < Yx) {
		uint Apos = time*total + istart;
		uint Bpos = wstart + _Yx;	//	Dot1d does not store W as Dot2d       in fact Dot2D.T = Dot1d  (it would be better to change it)

		float sum = 0;
		for (uint i=0; i < Ax; i++) {
			sum += var[Apos] * weight[Bpos];
			//printf("%i, %i, %f, %f", _Yx, i, var[Apos], weight[Bpos]);
			Apos++;
			Bpos += Yx;
		}
		sum += weight[wstart + Yx*Ax + _Yx];

		ACTIVATION_USE(activ, sum, sum);

		var[time*total + ystart + _Yx] = sum;

		//printf("%f\n", var[time*total + ystart + _Yx]);
	}
};