#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void dot1d_forward2_th11(
	uint Ax, uint Yx, uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart, uint l2start,
	uint total, uint wsize, uint lsize, uint l2size,
	float * var, float * weight, float * locd, float * locd2,
	uint sets)
{
	//	Toutes les arrays _dd_ sont relatives a dS/dw[i] (dw)
	
	uint x = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (x < Yx) {

		uint Apos;// = time*sets*total + set*total + input_start;
		uint Bpos;// + _Yx*Ax;

		//uint __seed;
		//float value;

		float sum = 0;
		for (uint i=0; i < Ax; i++) {

			Apos = time*sets*total + set*total + istart + i;
			Bpos = set*wsize + wstart + x + i*Yx;

			//__seed = Apos * (seed+1);

			//value = pseudo_randomf(__seed);

			//if ((value) >= drop_rate) {
			sum += var[Apos] * weight[Bpos];
			//}
		}
		sum += weight[set*wsize + wstart + Ax*Yx + x];
		
		float __locd, __locd2;

		ACTIVATION_FORWARD_2(activ, sum, sum, __locd, __locd2);

		var[time*sets*total + set*total + ystart + x] = sum;		//same assembler than putting it in if/else structure
		locd[time*sets*lsize + set*lsize + lstart + x] = __locd;
		locd2[time*sets*l2size + set*l2size + l2start + x] = __locd2;
	}
}