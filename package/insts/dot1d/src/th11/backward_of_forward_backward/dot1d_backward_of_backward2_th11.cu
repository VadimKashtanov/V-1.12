#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

//y = ax + b
//Backward:
//da += dy * x
//dx += dy * a
//db += dy

__global__
void dot1d_backward_of_backward2_th11(
	uint Ax, uint Yx, uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	float * _dd_var, float * _dd_weight, float * _dd_locd, float * _dd_grad, float * _dd_meand,
	uint sets)
{
	//	Toutes les arrays _dd_ sont relatives a dS/dw[i] (dw)
	
	/*	Kernel coordinates	*/
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (_Yx < Yx) {
		uint Apos = time*sets*total + set*total + istart;
		//uint weight_start = set*wsize + wstart;
		uint Bpos = set*wsize + wstart + _Yx;// _Yx*Ax;

		uint lpos = time*sets*lsize + set*lsize + lstart + _Yx;
		uint ypos = time*sets*total + set*total + ystart + _Yx;

		float dlds = locd[lpos] * grad[ypos];

		//meand[wstart + Yx*Ax + _Yx] += dlds;
		float D_dlds = _dd_meand[set*wsize + wstart + Yx*Ax + _Yx];

		float tmp;
		for (uint i=0; i < Ax; i++) {
			//atomicAdd(&grad[Apos], dlds * weight[Bpos]);
			tmp = _dd_grad[Apos];
			if (tmp != 0) {
				D_dlds += tmp * weight[Bpos];
				atomicAdd(&_dd_weight[Bpos], tmp * dlds);
			}

			//atomicAdd(&meand[Bpos], dlds * var[Apos]);
			tmp = _dd_meand[Bpos];
			if (tmp != 0) {
				D_dlds += tmp * var[Apos];
				atomicAdd(&_dd_var[Apos], tmp * dlds);
			}
			
			Apos++;
			Bpos += Yx;
		}

		//Backward of
		//float dlds = locd[time*sets*lsize + set*lsize + lstart + _Yx] * grad[time*sets*total + set*total + ystart + _Yx];
		if (D_dlds != 0) {
			atomicAdd(&_dd_grad[ypos], D_dlds * locd[lpos]);
			atomicAdd(&_dd_locd[lpos], D_dlds * grad[ypos]);
		}
	}
};