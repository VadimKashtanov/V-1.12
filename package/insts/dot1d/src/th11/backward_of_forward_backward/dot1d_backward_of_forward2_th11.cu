#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void dot1d_backward_of_forward2_th11(
	uint Ax, uint Yx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart, uint l2start,
	uint total, uint wsize, uint lsize, uint l2size,
	float * var, float * weight, float * locd, float * locd2, float * grad, float * meand,
	float * _dd_var, float * _dd_weight, float * _dd_locd, float * _dd_grad, float * _dd_meand,
	uint sets)
{
	//	Toutes les arrays _dd_ sont relatives a dS/dw[i] (dw)

	uint x = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (x < Yx) {

		uint Apos;// = time*sets*total + set*total + input_start;
		uint Bpos;// + _Yx*Ax;

		/*float sum = 0;
		for (uint i=0; i < Ax; i++) {

			Apos = time*sets*total + set*total + input_start + i;
			Bpos = set*wsize + wstart + x + i*Yx;

			//sum += var[Apos] * weight[Bpos];
		}
		sum += weight[set*wsize + wstart + Ax*Yx + x];
		
		float __locd, __locd2;

		var[time*sets*total + set*total + ystart + x] = sum;		//same assembler than putting it in if/else structure
		locd[time*sets*lsize + set*lsize + lstart + x] = __locd;
		locd2[time*sets*l2size + set*l2size + l2start + x] = __locd2;*/

		//__locd(s) est fonction de s, donc il faut derivee s. Voir le doc fondateur.
		//__locd2 est pas lié a forward_backward, il vient en appuis. On derive que le `var = sum` et `locd = __locd(s)`
		//car seul locd joue dans une des multiplication 

		uint ypos = time*sets*total + set*total + ystart + x;
		uint lpos = time*sets*lsize + set*lsize + lstart + x;
		uint l2pos = time*sets*l2size + set*l2size + l2start + x;

		float ds = 0;
		ds += locd2[l2pos] * _dd_locd[lpos];
		ds += locd[lpos] * _dd_var[ypos];

		atomicAdd(&_dd_weight[set*wsize + wstart + Ax*Yx + x], ds);

		if (ds != 0) {
			for (uint k=0; k < Ax; k++) {
				Apos = time*sets*total + set*total + istart + k;
				Bpos = set*wsize + wstart + x + k*Yx;

				//sum += var[Apos] * weight[Bpos];
				atomicAdd(&_dd_var[Apos], ds * weight[Bpos]);
				atomicAdd(&_dd_weight[Bpos], ds * var[Apos]);
			}
		}
	}
}