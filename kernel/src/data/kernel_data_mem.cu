#include "kernel/head/data.cuh"

Data_t * data_load(uint batchs, uint inputs, uint outputs, uint lines) {
	Data_t * ret = (Data_t*)malloc(sizeof(Data_t));

	ret->batchs = batchs;
	ret->inputs = inputs;
	ret->outputs = outputs;
	ret->lines = lines;

	ret->input = (float*)malloc(sizeof(float) * ret->lines * ret->inputs);
	ret->output = (float*)malloc(sizeof(float) * ret->lines * ret->outputs);

	ret->input_d = 0;
	ret->output_d = 0;

	return ret;
};

void data_cudamalloc(Data_t * data) {
	SAFE_CUDA(hipMalloc((void**)&data->input_d, sizeof(float) * data->lines * data->inputs));
	SAFE_CUDA(hipMalloc((void**)&data->output_d, sizeof(float) * data->lines * data->outputs));
};

void data_cudamemcpy(Data_t * data) {
	SAFE_CUDA(hipMemcpy(
		data->input_d,
		data->input,
		sizeof(float) * data->inputs * data->lines,
		hipMemcpyHostToDevice))

	SAFE_CUDA(hipMemcpy(
		data->output_d,
		data->output,
		sizeof(float) * data->outputs * data->lines,
		hipMemcpyHostToDevice))
};

void data_free_cudamalloc(Data_t * data) {
	if (data->input_d) SAFE_CUDA(hipFree(data->input_d));
	if (data->output_d) SAFE_CUDA(hipFree(data->output_d));
};

void data_free(Data_t * data) {
	if (data->input) free(data->input);
	if (data->output) free(data->output);

	data_free_cudamalloc(data);

	free(data);
};
