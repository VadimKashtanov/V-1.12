#include "kernel/head/use.cuh"

Use_t* use_mk(Mdl_t * mdl, Data_t * data) {
	Use_t * ret = (Use_t*)malloc(sizeof(Use_t));

	//	Dependances
	ret->mdl = mdl;
	ret->data = data;

	//	Weights
	SAFE_CUDA(hipMalloc((void**)&ret->weight_d, sizeof(float) * mdl->weights));
	SAFE_CUDA(hipMemcpy(ret->weight_d, mdl->weight, sizeof(float) * mdl->weights, hipMemcpyHostToDevice));

	//	Vars
	SAFE_CUDA(hipMalloc((void**)&ret->var_d, sizeof(float) * data->lines * mdl->total));
	//SAFE_CUDA(hipMemset(ret->var_d, 0, sizeof(float) * data->lines * mdl->total));	//vars have to be set and start from input

	return ret;
};

void use_free(Use_t * use) {
	SAFE_CUDA(hipFree(use->var_d));
	SAFE_CUDA(hipFree(use->weight_d));
	free(use);
};
