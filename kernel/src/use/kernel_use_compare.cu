#include "kernel/head/use.cuh"

void use_compare_weights(Use_t * use, float * with_this) {
	float * tmpt = (float*)malloc(sizeof(float) * (use->mdl->weights));
	SAFE_CUDA(hipMemcpy(tmpt, use->weight_d, sizeof(float) * (use->mdl->weights), hipMemcpyDeviceToHost));

	int lbl;

	for (uint i=0; i < use->mdl->weights; i++) {
		lbl = find_sep(use->mdl->wsep, i);

		if (lbl != -1)
			printf("|| (%i) %s\n", i, use->mdl->wsep->labels[lbl]);

		if (compare_floats(tmpt[i], with_this[i], COMPARE_DEEPH)) 
			printf("|| %i |  \033[42m %f --- %f \033[0m \n", i, tmpt[i], with_this[i]);
		else
			printf("|| %i |  \033[41m %f --- %f \033[0m \n", i, tmpt[i], with_this[i]);
	}

	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};

void use_compare_vars(Use_t * use, float * with_this) {
	float * tmpt = (float*)malloc(sizeof(float) * (use->data->lines * use->mdl->total));
	SAFE_CUDA(hipMemcpy(tmpt, use->var_d, sizeof(float) * (use->data->lines * use->mdl->total), hipMemcpyDeviceToHost));

	int lbl;
	uint pos;

	for (uint l=0; l < use->data->lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint i=0; i < use->mdl->total; i++) {
			lbl = find_sep(use->mdl->vsep, i);

			if (lbl != -1) {
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("|| (%i) %s\n", i, use->mdl->vsep->labels[lbl]);
			}

			pos = l*use->mdl->total + i;
				
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				
			if (compare_floats(tmpt[pos], with_this[pos], COMPARE_DEEPH)) 
				printf("|| %i |  \033[42m %f --- %f \033[0m \n", i, tmpt[pos], with_this[pos]);
			else
				printf("|| %i |  \033[41m %f --- %f \033[0m \n", i, tmpt[pos], with_this[pos]);
		}
	}

	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};
