#include "kernel/head/use.cuh"

void use_print_weights(Use_t * use) {
	float * tmpt = (float*)malloc(sizeof(float) * (use->mdl->weights));
	SAFE_CUDA(hipMemcpy(tmpt, use->weight_d, sizeof(float) * use->mdl->weights, hipMemcpyDeviceToHost));

	int lbl;
	for (uint i=0; i < use->mdl->weights; i++) {
		lbl = find_sep(use->mdl->wsep, i);

		if (lbl != -1)
			printf("|| (%i) %s\n", i, use->mdl->wsep->labels[lbl]);

		printf("|| %i |  \033[93m %f \033[0m \n", i, tmpt[i]);
	}

	free(tmpt);
};

void use_print_vars(Use_t * use) {
	float * tmpt = (float*)malloc(sizeof(float) * (use->data->lines * use->mdl->total));
	SAFE_CUDA(hipMemcpy(tmpt, use->var_d, sizeof(float) * (use->data->lines * use->mdl->total), hipMemcpyDeviceToHost));

	int lbl;
	uint pos;

	for (uint l=0; l < use->data->lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint i=0; i < use->mdl->total; i++) {
			lbl = find_sep(use->mdl->vsep, i);

			if (lbl != -1) {
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("|| (%i) %s\n", i, use->mdl->vsep->labels[lbl]);
			}

			pos = l*use->mdl->total + i;
				
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				
			printf("|| %i |  \033[93m %f \033[0m \n", i, tmpt[pos]);
		}
	}

	free(tmpt);
};

//------------------

void use_print_all(Use_t * use) {
	use_print_weights(use);
	use_print_vars(use);
};
