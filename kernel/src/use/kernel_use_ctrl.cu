#include "kernel/head/use.cuh"

void use_set_input(Use_t * use) {
	for (uint t=0; t < use->data->lines; t++) {
		SAFE_CUDA(
			hipMemcpy(
				use->var_d + t*use->mdl->total,
				use->data->input_d + t*use->mdl->inputs,
				sizeof(float) * use->mdl->inputs,
				hipMemcpyDeviceToDevice
			)
		)
	}
};

void use_forward(Use_t * use) {
	for (uint t=0; t < use->data->lines; t++)
		for (uint i=0; i < use->mdl->insts; i++)
			INST_USE[use->mdl->inst[i]->id](use, use->mdl->inst[i], t);
};