#include "hip/hip_runtime.h"
#include "kernel/head/etc.cuh"

FILE * SAFE_FOPEN(const char * file, const char * mode) {
	FILE * fp = fopen(file, mode);
	if (fp == 0)
		ERR("Le fichier %s n'existe pas.", file);
	return fp;
};

uint read_uint(FILE * fp) {
	uint _123;
	fread(&_123, sizeof(uint), 1, fp);
	return _123;
};

uint read_123(FILE * fp) {
	uint _123;
	fread(&_123, sizeof(uint), 1, fp);
	return (uint)(_123 == 123);
};

void write_123(FILE * fp) {
	uint _123 = 123;
	fwrite(&_123, sizeof(uint), 1, fp);
};

//#define is_123(fp) do {if (read_uint(fp) != 123) ERR("Etait attendu 123");}while(0);
void is_123(FILE * fp) {
	uint _123 = read_uint(fp);

	if (_123 != 123) {
		float c;
		memcpy(&c, &_123, sizeof(float));
		ERR("Etait attendu 123. Obtenu : (uint)%i, (float)%f", _123, (float)c);
	}
}

/*void etc_parse_arguments(uint argc, char ** argv, uint paramc, const char ** paramv, char ** correspondance) {
	for (uint i=0; i < argc; i++) {
		//	Find the correspondance
		for (uint j=0; j < paramc; j++) {
			if (strcmp(argv[i+1], paramv[j]) == 0) {	//'-sets 3' to skip the '-'
				correspondance[j] = argv[i];
				i++;
				break;
			}
		}

	}
};*/

static void terminale_plateau_pixels(char ** premier_text, bool ** pixs, char ** dig0s, char ** dig1s, uint len, uint H) {
	//
	uint _max = strlen(premier_text[0]);
	for (uint i=0; i < H; i++)
		if (strlen(premier_text[i]) > _max) _max = strlen(premier_text[i]);
	
	//
	for (uint y=0; y < H; y++) {
		printf("%s", premier_text[y]);
		for (uint j=0; j < (1+_max-strlen(premier_text[y])); j++) printf(" ");
		//
		printf("\033[100;94;4m ");
		for (uint i=0; i < len; i++)
			printf("\033[%i;94;4m%c%c\033[100;4m ", (pixs[y][i]== 0 ? 100 : 107), dig0s[y][i], dig1s[y][i]);
		printf("\033[0m\n");
	}
};

void term_plot(float * values, uint len, uint H) {
	if (len > 100 || len == 0)
		ERR("La taille d'un array doit etre entre 1 et 100. Ici elle est de %i", len);

	if (H <= 1)
		ERR("H doit etre > 1");

	uint h = H - 1;
	printf("Plotting [%.5g", values[0]);
	for (uint i=1; i < len; i++) printf(", %.5g", values[i]);
	printf("]\n");

	//	Valeur max et min
	float _min = values[0];
	float _max = values[0];
	for (uint i=1; i < len; i++) {
		if (values[i] > _max) _max = values[i];
		if (values[i] < _min) _min = values[i];
	}

	//	Maintenant valeur entre 0.0 et 1.0
	float norm_values[len];
	for (uint i=0; i < len; i++)
		norm_values[i] = (values[i] - _min)/(_max - _min);

	//	Construction des digits et texts premier
	char ** premier_text = (char**)malloc(sizeof(char*) * H);
	bool ** pixs = (bool**)malloc(sizeof(bool*) * H);
	char ** dig0s = (char**)malloc(sizeof(char*) * H);
	char ** dig1s = (char**)malloc(sizeof(char*) * H);

	for (uint y=0; y < H; y++) {
		pixs[y] = (bool*)malloc(sizeof(bool) * len);
		dig0s[y] = (char*)malloc(sizeof(char) * len);
		dig1s[y] = (char*)malloc(sizeof(char) * len);
		for (uint x=0; x < len; x++) {
			dig0s[y][x] = ' ';
			dig1s[y][x] = ' ';
			pixs[y][x] = 0;
		}
	}

	uint hauteur;
	for (uint i=0; i < len; i++) {
		hauteur = (uint)(H - (norm_values[i]*h) - 1);

		dig0s[hauteur][i] = 48 + (uint)roundf((i - (i%10))/10);
		dig1s[hauteur][i] = 48 + i%10;
		for (uint k=hauteur; k < H; k++) {
			pixs[k][i] = 1;
		}
	};

	uint _strlen;
	float val;
	for (uint y=0; y < H; y++) {
		val = (_max - (_max-_min)*y/h);
		_strlen = snprintf(NULL, 0, "%.5g", val);
		premier_text[y] = (char*)malloc(_strlen + 1);
		snprintf(premier_text[y], _strlen + 1, "%f", val);
	};

	terminale_plateau_pixels(premier_text, pixs, dig0s, dig1s, len, H);

	for (uint i=0; i < H; i++)
		free(premier_text[i]);
	free(premier_text);

	for (uint y=0; y < H; y++) {
		free(pixs[y]);
		free(dig0s[y]);
		free(dig1s[y]);
	}
	free(pixs);
	free(dig0s);
	free(dig1s);
};

//=============================================================================================

float cuda_get_float(float * arr_d, uint pos) {
	float res;
	SAFE_CUDA(hipMemcpy(&res, arr_d + pos, sizeof(float) * 1, hipMemcpyDeviceToHost));
	return res;
};

uint cuda_get_uint(uint * arr_d, uint pos) {
	uint res;
	SAFE_CUDA(hipMemcpy(&res, arr_d + pos, sizeof(uint) * 1, hipMemcpyDeviceToHost));
	return res;
};