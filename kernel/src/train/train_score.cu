#include "kernel/head/train.cuh"

void train_solo_compute_score(Train_t * train, uint start_seed) {
	//
	train_set_input(train);
	train_forward(train, start_seed);

	//	Loss
	score_loss(train);
	score_score(train);
};

static void ordoner_les_scores(Train_t * train) {
	uint c;
	for (uint i=0; i < train->sets; i++) train->set_rank[i] = i;
	for (uint i=0; i < train->sets; i++) {
		for (uint j=0; j < train->sets; j++) {
			if (train->set_score[train->set_rank[i]] < train->set_score[train->set_rank[j]]) {
				c = train->set_rank[i];
				train->set_rank[i] = train->set_rank[j];
				train->set_rank[j] = c;
			}
		}
	}

	SAFE_CUDA(hipMemcpy(train->set_rank_d, train->set_rank, sizeof(uint)*train->sets, hipMemcpyHostToDevice));
	SAFE_CUDA(hipMemcpy(train->set_score_d, train->set_score, sizeof(float)*train->sets, hipMemcpyHostToDevice));
}

void score_score(Train_t * train) {
	SCORE_SCORE[train->score->id](train);	//calcule train->set_score a partire de loss (il faut obligatoirement faire loss avant)

	ordoner_les_scores(train);
};

void score_loss(Train_t * train) {
	SCORE_LOSS[train->score->id](train);
};

void score_dloss(Train_t * train) {
	SCORE_DLOSS[train->score->id](train);
};

void score_ddloss(Train_t * train) {
	SCORE_DDLOSS[train->score->id](train);
};