#include "kernel/head/train.cuh"

void train_null_for_dS(Train_t * train) {
	//SAFE_CUDA(hipMemset(train->_var, 0, sizeof(float) * train->sets * train->data->lines * train->mdl->vars));
	SAFE_CUDA(hipMemset(train->_meand, 0, sizeof(float) * train->sets * train->mdl->weights))
	SAFE_CUDA(hipMemset(train->_grad, 0, sizeof(float) * train->sets * train->data->lines * train->mdl->total))
};

void train_forward(Train_t * train, uint start_seed) {
	for (uint t=0; t < train->data->lines; t++) {
		for (uint i=0; i < train->mdl->insts; i++) {
			INST_FORWARD[train->mdl->inst[i]->id](train, train->mdl->inst[i], t, start_seed);
		}
	}
};

void train_backward(Train_t * train, uint start_seed) {
	for (int t=train->data->lines-1; t >= 0; t--) {
		for (int i=train->mdl->insts-1; i >= 0; i--) {
			INST_BACKWARD[train->mdl->inst[i]->id](train, train->mdl->inst[i], t, start_seed);
		}
	}
};
