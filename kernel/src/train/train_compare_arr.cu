#include "kernel/head/train.cuh"

static uint print_arr_dim2(
	Separators_t * sep, float * arr, float * arr1,
	const char * dim0_str, //char * dim1_str,
	uint dim0, uint dim1,
	float tolerance)
{
	int lbl;
	uint pos;

	uint erreurs = 0;

	for (uint d0=0; d0 < dim0; d0++) {
		printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
		printf("%s #%i ============= \n", dim0_str, d0);
		for (uint d1=0; d1 < dim1; d1++) {
			lbl = find_sep(sep, d1);

			if (lbl != -1) {
				printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
				printf("|| (%i) %s\n", d1, sep->labels[lbl]);
			}

			pos = d0*dim1 + d1;
			
			printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
			
			if (compare_floats(arr[pos], arr1[pos], tolerance)) {
				printf("|| %i |  \033[42m %f --- %f \033[0m \n", d1, arr[pos], arr1[pos]);
			} else {
				printf("|| %i |  \033[41m %f --- %f \033[0m \n", d1, arr[pos], arr1[pos]);
				erreurs = 1;
			}
		}
	}

	printf("        c/cuda ---- python\n");

	return erreurs;
};

static uint print_arr_dim3(
	Separators_t * sep, float * arr, float * arr1,
	const char * dim0_str, const char * dim1_str, //char * dim2_str,
	uint dim0, uint dim1, uint dim2,
	float tolerance)
{
	int lbl;
	uint pos;

	uint erreurs = 0;

	for (uint d0=0; d0 < dim0; d0++) {
		printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));
		printf("%s = %i ################### \n", dim0_str, d0);
		for (uint d1=0; d1 < dim1; d1++) {
			printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
			printf("\033[%im||\033[0m", (d1 % 2 ? 93 : 96)); // '||' du set
			printf("%s #%i ============= \n", dim1_str, d1);
			for (uint d2=0; d2 < dim2; d2++) {
				lbl = find_sep(sep, d2);

				if (lbl != -1) {
					printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
					printf("\033[%im||\033[0m", (d1 % 2 ? 93 : 96)); // '||' du set
					printf("|| (%i) %s\n", d2, sep->labels[lbl]);
				}

				pos = d0*dim1*dim2 + d1*dim2 + d2;
				
				printf("\033[%im||\033[0m", (d0 % 2 ? 92 : 91));	// '||' de la ligne
				printf("\033[%im||\033[0m", (d1 % 2 ? 93 : 96)); // '||' du set
				
				if (compare_floats(arr[pos], arr1[pos], tolerance)) {
					printf("|| %i |  \033[42m %f --- %f \033[0m \n", d2, arr[pos], arr1[pos]);
				} else {
					printf("|| %i |  \033[41m %f --- %f \033[0m \n", d2, arr[pos], arr1[pos]);
					erreurs = 1;
				}
			}
		}
	}

	printf("         c/cuda ---- python\n");

	return erreurs;
};

static const char * arrs[11] = {
	"_weight", "_var", "_locd", "_grad", "_meand", "_dd_weight", "_dd_var", "_dd_locd", "_dd_grad", "_dd_meand", "_locd2"
};

static int find(const char * nom) {
	for (uint i=0; i < 11; i++)
		if (strcmp(nom, arrs[i]) == 0)
			return i;
	return -1;
};

uint train_eq_arr(Train_t * train, const char * nom, float * cpu_arr, float tolerance) {
	Mdl_t * mdl = train->mdl;

	uint sets = train->sets;
	uint lines = train->lines;
	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint lsize = mdl->locds;
	uint l2size = mdl->locd2s;

	uint dws = train->dws;

	uint arrs_tens[11][3] = {
		{sets, wsize, 0},
		{lines, sets, total},
		{lines, sets, lsize},

		{lines, sets, total},
		{sets, wsize, 0},

		{dws, sets, wsize},
		{lines, sets, total},
		{lines, sets, lsize},
		{lines, sets, total},
		{sets, wsize, 0},
		{lines, sets, l2size}
	};

	float * arrs_ptrs[11] = {
		train->_weight, train->_var, train->_locd,
		train->_grad, train->_meand,
		train->_dd_weight, train->_dd_var, train->_dd_locd, train->_dd_grad, train->_dd_meand, train->_locd2
	};

	int pos = find(nom);

	if (pos == -1) ERR("train->%s n'existe pas", nom);
	if (arrs_ptrs[pos] == 0) ERR("train->%s == 0", nom);

	uint len = arrs_tens[pos][0] * arrs_tens[pos][1] * (arrs_tens[pos][2] == 0 ? 1 : arrs_tens[pos][2]);

	float * tmp_cpu = (float*)malloc(sizeof(float) * len);
	SAFE_CUDA(hipMemcpy(tmp_cpu, arrs_ptrs[pos], sizeof(float) * len, hipMemcpyDeviceToHost));

	for (uint i=0; i < len; i++)
		if (compare_floats(tmp_cpu[i], cpu_arr[i], tolerance) != 1)
			return 0;

	free(tmp_cpu);

	return 1;
};

void train_print_compare_arr(Train_t * train, const char * nom, float * cpu_arr, float tolerance) {
	Mdl_t * mdl = train->mdl;

	uint sets = train->sets;
	uint lines = train->lines;
	uint total = mdl->total;
	uint wsize = mdl->weights;
	uint lsize = mdl->locds;
	uint l2size = mdl->locd2s;

	uint dws = train->dws;

	Separators_t * wsep = mdl->wsep, * vsep = mdl->vsep, * lsep = mdl->lsep, * l2sep = mdl->l2sep;

	uint arrs_dim[11] = {
		2, 3, 3, 3, 2, 3, 3, 3, 3, 3, 2
	};

	uint arrs_tens[11][3] = {
		{sets, wsize, 0},
		{lines, sets, total},
		{lines, sets, lsize},

		{lines, sets, total},
		{sets, wsize, 0},

		{dws, sets, wsize},
		{lines, sets, total},
		{lines, sets, lsize},
		{lines, sets, total},
		{sets, wsize, 0},
		{lines, sets, l2size}
	};

	const char * arr_tens_lbl[11][2] = {
		{"Set", 0},
		{"Line", "Set"},
		{"Line", "Set"},

		{"Line", "Set"},
		{"Set", 0},

		{"DW", "Set"},
		{"Line", "Set"},
		{"Line", "Set"},
		{"Line", "Set"},
		{"Set", 0},
		{"Line", "Set"}
	};

	float * arrs_ptrs[11] = {
		train->_weight, train->_var, train->_locd,
		train->_grad, train->_meand,
		train->_dd_weight, train->_dd_var, train->_dd_locd, train->_dd_grad, train->_dd_meand, train->_locd2
	};

	Separators_t * arrs_sep[11] = {
		wsep, vsep, lsep,
		vsep, wsep,
		wsep, vsep, lsep, vsep, wsep, l2sep
	};

	int pos = find(nom);

	if (pos == -1) ERR("train->%s n'existe pas", nom);
	if (arrs_ptrs[pos] == 0) ERR("train->%s == 0", nom);

	uint len = arrs_tens[pos][0] * arrs_tens[pos][1] * (arrs_tens[pos][2] == 0 ? 1 : arrs_tens[pos][2]);

	float * train_cpu = (float*)malloc(sizeof(float) * len);
	SAFE_CUDA(hipMemcpy(train_cpu, arrs_ptrs[pos], sizeof(float) * len, hipMemcpyDeviceToHost));

	if (arrs_dim[pos] == 2) {
		print_arr_dim2(arrs_sep[pos], train_cpu, cpu_arr,
			arr_tens_lbl[pos][0],
			arrs_tens[pos][0], arrs_tens[pos][1],
			tolerance
		);
	} else {
		print_arr_dim3(arrs_sep[pos], train_cpu, cpu_arr,
			arr_tens_lbl[pos][0], arr_tens_lbl[pos][1],
			arrs_tens[pos][0], arrs_tens[pos][1], arrs_tens[pos][2],
			tolerance
		);
	}
	free(train_cpu);
};