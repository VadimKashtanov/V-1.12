#include "hip/hip_runtime.h"
#include "kernel/head/train.cuh"

static __global__ void kernel_set_input(float * _var, float * _input, uint total, uint sets, uint inputs, uint lines) {
	uint _inp = threadIdx.x + blockIdx.x * blockDim.x,	\
		 line = threadIdx.y + blockIdx.y * blockDim.y,	\
		 set = blockIdx.z;

	if (_inp < inputs && line < lines) {
		_var[line*sets*total + set*total + _inp] = _input[line*inputs + _inp];
	}
};

void train_set_input(Train_t * train) {
	kernel_set_input<<<dim3(KERN_DIV(train->mdl->inputs,32), KERN_DIV(train->data->lines,32), train->sets),dim3(32,32,1)>>>(
		train->_var, train->data->input_d, train->mdl->total, train->sets, train->mdl->inputs, train->data->lines);
	SAFE_CUDA(hipPeekAtLastError());
};

void train_restart(Train_t * train) {
	SAFE_CUDA(hipMemset(train->_var, 0, sizeof(float) * train->sets * train->mdl->weights));
};