#include "kernel/head/train.cuh"

Train_t* mk_train(
	Mdl_t * mdl, Data_t * data,
	const Config_t * score, const Config_t * opti, const Config_t * gtic,
	const uint calcule_d,
	const uint calcule_dd,
	const uint dws, const uint * dw)
{
	Train_t * ret = (Train_t*)malloc(sizeof(Train_t));

	//	Mdl & Data
	ret->mdl = mdl;
	ret->data = data;
	ret->lines = data->lines;

	if (mdl->inputs != data->inputs) ERR("Mdl->inputs (%i) != Data->inputs (%i)", mdl->inputs, data->inputs);
	if (mdl->outputs != data->outputs) ERR("Mdl->outputs (%i) != Data->outputs (%i)", mdl->outputs, data->outputs);

	if (!mdl->capable_df && calcule_d) ERR("mdl->capable_df (%i) != calcule_dF (%i)", mdl->capable_df, calcule_d);
	if (!mdl->capable_ddf && calcule_dd) ERR("mdl->capable_ddf (%i) != calcule_ddF (%i)", mdl->capable_ddf, calcule_dd);

	//	Sets & calcule_dd
	ret->sets = 0;
	ret->calcule_d = calcule_d;
	ret->calcule_dd = calcule_dd;
	ret->dws = dws;
	ret->dw = (uint*)malloc(sizeof(uint) * dws);
	memcpy(ret->dw, dw, sizeof(uint) * dws);
	
	//	Score
	ret->score = cpy_config(score);
	SCORE_MK[score->id](ret);

	//	Opti
	ret->opti = cpy_config(opti);
	OPTI_MK[opti->id](ret);

	//	Gtic
	ret->gtic = cpy_config(gtic);
	GTIC_MK[gtic->id](ret);

	if (ret->sets == 0) ERR("Le Gtic devait donner un nombre de sets non null");
	uint sets = ret->sets;

	uint lines = data->lines;
	uint ws = mdl->weights;

	SAFE_CUDA(hipMalloc((void**)&ret->_weight, sizeof(float) * (ws*sets)));
	SAFE_CUDA(hipMalloc((void**)&ret->_var, sizeof(float) * (mdl->total*sets*lines)));

	if (calcule_d) SAFE_CUDA(hipMalloc((void**)&ret->_locd, sizeof(float) * (mdl->locds*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_grad, sizeof(float) * (mdl->total*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_meand, sizeof(float) * (ws*sets)));

	if (calcule_dd) {
		SAFE_CUDA(hipMalloc((void**)&ret->_locd2, sizeof(float) * (mdl->locd2s * sets * lines)));
		SAFE_CUDA(hipMalloc((void**)&ret->_dd_weight, sizeof(float) * (dws * sets*ws)));
		SAFE_CUDA(hipMalloc((void**)&ret->_dd_var, sizeof(float) * (mdl->total*sets*lines)));
		SAFE_CUDA(hipMalloc((void**)&ret->_dd_locd, sizeof(float) * (mdl->locds*sets*lines)));
		SAFE_CUDA(hipMalloc((void**)&ret->_dd_grad, sizeof(float) * (mdl->total*sets*lines)));
		SAFE_CUDA(hipMalloc((void**)&ret->_dd_meand, sizeof(float) * (ws*sets)));
	}

	if (!calcule_d) ret->_locd = 0;
	
	if (!calcule_dd) {
		ret->_locd2 = 0;
		ret->_dd_weight = 0;
		ret->_dd_var = 0;
		ret->_dd_grad = 0;
		ret->_dd_locd = 0;
		ret->_dd_meand = 0;
	}

	//
	ret->set_score = (float*)malloc(sizeof(float) * sets);
	SAFE_CUDA(hipMalloc((void**)&ret->set_score_d, sizeof(float) * sets));
	
	ret->set_rank = (uint*)malloc(sizeof(uint) * sets);
	SAFE_CUDA(hipMalloc((void**)&ret->set_rank_d, sizeof(uint) * sets));

	ret->podium = (uint*)malloc(sizeof(uint) * sets);

	return ret;
};

Config_t * score_mk_config(uint id) {
	return config_mk(id, SCORE_params[id], SCORE_defaults[id]);
};

Config_t * opti_mk_config(uint id) {
	return config_mk(id, OPTI_params[id], OPTI_defaults[id]);
};

Config_t * gtic_mk_config(uint id) {
	return config_mk(id, GTIC_params[id], GTIC_defaults[id]);
};

void train_free(Train_t * ret) {
	free(ret->dw);
	
	SAFE_CUDA(hipFree(ret->_weight));
	SAFE_CUDA(hipFree(ret->_var));
	
	if (ret->calcule_d == 1) SAFE_CUDA(hipFree(ret->_locd));

	SAFE_CUDA(hipFree(ret->_grad));
	SAFE_CUDA(hipFree(ret->_meand));
	
	if (ret->calcule_dd == 1) {
		SAFE_CUDA(hipFree(ret->_locd2));
		SAFE_CUDA(hipFree(ret->_dd_weight));
		SAFE_CUDA(hipFree(ret->_dd_var));
		SAFE_CUDA(hipFree(ret->_dd_locd));
		SAFE_CUDA(hipFree(ret->_dd_grad));
		SAFE_CUDA(hipFree(ret->_dd_meand));
	}

	SAFE_CUDA(hipFree(ret->set_score_d));
	SAFE_CUDA(hipFree(ret->set_rank_d));
	free(ret->set_rank);
	free(ret->set_score);
	free(ret->podium);
	
	GTIC_FREE[ret->gtic->id](ret);
	OPTI_FREE[ret->opti->id](ret);
	SCORE_FREE[ret->score->id](ret);

	config_free(ret->score);
	config_free(ret->opti);
	config_free(ret->gtic);

	free(ret);
};