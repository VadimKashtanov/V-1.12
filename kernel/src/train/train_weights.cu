#include "hip/hip_runtime.h"
#include "kernel/head/train.cuh"

//	---------------------------------------------------------------------------------------------
//	------------------------------------ Aleatoire ----------------------------------------------
//	---------------------------------------------------------------------------------------------	

static __global__ void kernel_random_weights(uint rnd_seed, uint weights, float * _weight) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights) {
		_weight[pos] = pseudo_randomf_minus1_1(rnd_seed + pos);
	}
};

void train_random_weights(Train_t * train, uint rnd_seed) {
	kernel_random_weights<<<dim3(KERN_DIV(train->mdl->weights,32), train->sets), dim3(32,1)>>>(
		rnd_seed, train->mdl->weights, train->_weight);
	CUDA_WAIT_KER()
};

//	---------------------------------------------------------------------------------------------
//	----------------------------- Random Weight From Mdl ----------------------------------------
//	---------------------------------------------------------------------------------------------	

static __global__ void kernel_random_weights_from_mdl(uint rnd_seed, uint weights, float * _weight, float * mdl_weight_d, float coef) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights) {
		uint a = rnd_seed + pos;
		_weight[pos] = mdl_weight_d[wid] + coef*pseudo_randomf_minus1_1(a);
	}
};

void train_random_weights_from_mdl(Train_t * train, uint rnd_seed, float coef) {
	float * mdl_weights_d;
	SAFE_CUDA(hipMalloc((void**)&mdl_weights_d, sizeof(float)*train->mdl->weights));
	SAFE_CUDA(hipMemcpy(mdl_weights_d, train->mdl->weight, sizeof(float)*train->mdl->weights, hipMemcpyHostToDevice));

	kernel_random_weights_from_mdl<<<dim3(KERN_DIV(train->mdl->weights, 32), train->sets),dim3(32,1)>>>(
		rnd_seed, train->mdl->weights, train->_weight, mdl_weights_d, coef);	//coef = 0.1 c'est pas mal
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());

	SAFE_CUDA(hipFree(mdl_weights_d));
};

//	---------------------------------------------------------------------------------------------
//	----------------------- Injecter Parametres vers un set -------------------------------------
//	---------------------------------------------------------------------------------------------

void train_inject_weight_from_mdl_to_one_set(Train_t * train, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->_weight + set*train->mdl->weights, train->mdl->weight,
		sizeof(float)*train->mdl->weights, hipMemcpyHostToDevice));
};

void train_inject_weight_cpu_to_one_set(Train_t * train, float * weight, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->_weight + set*train->mdl->weights, weight,
		sizeof(float)*train->mdl->weights, hipMemcpyHostToDevice));
};

void train_inject_weight_gpu_to_one_set(Train_t * train, float * weight_d, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->_weight + set*train->mdl->weights, weight_d,
		sizeof(float)*train->mdl->weights, hipMemcpyDeviceToDevice));
};

//	---------------------------------------------------------------------------------------------
//	------------------------------ Train->weight -> Train->mdl ----------------------------------
//	---------------------------------------------------------------------------------------------

void train_cpy_ws_to_mdl(Train_t * train, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->mdl->weight, train->_weight + set*train->mdl->weights,
		sizeof(float)*train->mdl->weights, hipMemcpyDeviceToHost));
};