#include "kernel/head/train.cuh"

void train_ddS_first_null(Train_t * train) {
	Mdl_t * mdl = train->mdl;

	//SAFE_CUDA(hipMemset(train->_var, 0, sizeof(float) * train->sets * train->data->lines * mdl->total));
	SAFE_CUDA(hipMemset(train->_meand, 0, sizeof(float) * train->sets * mdl->weights));
	SAFE_CUDA(hipMemset(train->_grad, 0, sizeof(float) * train->sets * train->data->lines * mdl->total));
	//SAFE_CUDA(hipMemset(train->_locd, 0, sizeof(float) * train->sets * train->data->lines * mdl->locds));
	//SAFE_CUDA(hipMemset(train->_locd2, 0, sizeof(float) * train->sets * train->data->lines * mdl->locds2));
};

void train_forward2(Train_t * train, uint start_seed) {
	for (uint t=0; t < train->data->lines; t++) {
		for (uint i=0; i < train->mdl->insts; i++) {
			INST_FORWARD2[train->mdl->inst[i]->id](train, train->mdl->inst[i], t, start_seed);
		}
	}
};

void train_backward2(Train_t * train, uint start_seed) {
	for (int t=train->data->lines-1; t >= 0; t--) {
		for (int i=train->mdl->insts-1; i >= 0; i--) {
			INST_BACKWARD2[train->mdl->inst[i]->id](train, train->mdl->inst[i], t, start_seed);
		}
	}
};

void train_ddS_second_dd_restart(Train_t * train, uint dw) {
	Mdl_t * mdl = train->mdl;
	
	SAFE_CUDA(hipMemset(train->_dd_weight + dw*train->sets * mdl->weights, 0, sizeof(float) * train->sets * mdl->weights));	//tous vont etre memset
	SAFE_CUDA(hipMemset(train->_dd_var, 0, sizeof(float) * train->sets * train->data->lines * mdl->total));
	SAFE_CUDA(hipMemset(train->_dd_meand, 0, sizeof(float) * train->sets * mdl->weights));
	SAFE_CUDA(hipMemset(train->_dd_grad, 0, sizeof(float) * train->sets * train->data->lines * mdl->total));
	SAFE_CUDA(hipMemset(train->_dd_locd, 0, sizeof(float) * train->sets * train->data->lines * mdl->locds));

	float _1 = 1.0;
	//_dd_meand[i] = 1;
	for (uint set=0; set < train->sets; set++)
		SAFE_CUDA(hipMemcpy(train->_dd_meand + set*train->mdl->weights + train->dw[dw], &_1, sizeof(float) * 1, hipMemcpyHostToDevice));
};

void train_backward_of_backward2(Train_t * train, uint dw, uint start_seed) {
	for (uint ligne=0; ligne < train->data->lines; ligne++) {
		for (uint inst=0; inst < train->mdl->insts; inst++) {
			INST_BACKWARD_OF_BACKWARD2[train->mdl->inst[inst]->id](train, train->mdl->inst[inst], dw, ligne, start_seed);
		}
	}
};

void train_backward_of_forward2(Train_t * train, uint dw, uint start_seed) {
	for (int ligne=train->data->lines-1; ligne >= 0; ligne--) {
		for (int inst=train->mdl->insts-1; inst >= 0; inst--) {
			INST_BACKWARD_OF_FORWARD2[train->mdl->inst[inst]->id](train, train->mdl->inst[inst], dw, ligne, start_seed);
		}
	}
};