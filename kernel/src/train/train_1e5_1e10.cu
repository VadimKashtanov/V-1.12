#include "hip/hip_runtime.h"
#include "kernel/head/train.cuh"

static void __global__ _add_x_1e5(
	uint w, uint ws, uint sets,
	float * _weight,
	float changement)
{
	uint set = threadIdx.x + blockIdx.x * blockDim.x;
	if (set < sets)
		_weight[set*ws + w] += changement;
};

static void __global__ _dSdw_1e5(
	uint w, uint ws,
	float * _meand,
	float * score_1e5, float * score,
	uint sets)
{
	uint set = threadIdx.x + blockIdx.x * blockDim.x;
	if (set < sets)
		_meand[set*ws + w] = 1e5 * (score_1e5[set] - score[set]);
};

//	dS = F(x+1e-5)-F(x)
void train_dSdw_1e5(Train_t * train, uint start_seed) {
	uint ws = train->mdl->weights;
	uint sets = train->sets;

	float * _score_plus_1e5_d;
	SAFE_CUDA(hipMalloc((void**)&_score_plus_1e5_d, sizeof(float)*sets));
	for (uint w=0; w < ws; w++) {
		//	F(x+1e5)
		_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
			w, ws, sets,
			train->_weight,
			1e5);
		train_solo_compute_score(train, start_seed);
		SAFE_CUDA(hipMemcpy(_score_plus_1e5_d, train->set_score_d, float(float)*sets, hipMemcpyDeviceToDevice));
		_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
			w, ws, sets,
			train->_weight,
			-1e5);

		//	F(x+1e5)
		train_solo_compute_score(train, start_seed);

		//	f(x+)-f(x)
		_dSdw_1e5<<<dim3(KERN_DIV(train->sets, 32)),dim3(32)>>>(
			w, train->mdl->weights,
			train->_meand,
			_score_plus_1e5_d, train->set_score_d,
			train->sets
		);
		CUDA_WAIT_KER()
	}
	SAFE_CUDA(hipFree(_score_plus_1e5_d));
};

//===============================================================================
//===============================================================================
//===============================================================================

static void __global__ _dSdwdw_1e5(
	uint dw, uint ws,
	float * _dd_weight,
	float * _meand_1e5, float * _meand,
	uint sets)
{
	uint w = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = threadIdx.y + blockIdx.y * blockDim.y;
	if (w < ws && set < sets)
		_dd_weight[dw*sets*ws + set*ws + w] = 1e5 * (_meand_1e5[set] - _meand[set]);
};

//	ddS = dF(x+1e-5)-dF(x)
void train_dSdwdw_1e5(Train_t * train, uint start_seed) {
	uint ws = train->mdl->weights;
	uint sets = train->sets;
	uint dws = train->dws;

	float * _meand_1e5;
	SAFE_CUDA(hipMalloc((void**)&_meand_1e5, sizeof(float)*sets*ws));
	for (uint dw=0; dw < dws; dw++) {
		//	dF(x+1e-5)
		_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
			train->dw[dw], ws, sets,
			train->_weight,
			1e5);
		train_forward_backward(train, start_seed);
		SAFE_CUDA(hipMemcpy(_meand_1e5, train->_meand, sizeof(float)*ws*sets, hipMemcpyDeviceToDevice));
		_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
			train->dw[dw], ws, sets,
			train->_weight,
			-1e5);

		//	dF(x)
		train_forward_backward(train, start_seed);

		//	df(x+)-df(x)
		_dSdwdw_1e5<<<dim3(KERN_DIV(ws, 32), KERN_DIV(sets, 32)),dim3(32,32)>>>(
			train->dw[dw], ws,
			train->_dd_weight,
			_meand_1e5, train->_meand,
			train->sets
		);
		CUDA_WAIT_KER()
	}
	SAFE_CUDA(hipFree(_meand_1e5));
};

//===============================================================================
//===============================================================================
//===============================================================================

static void __global__ _dSdwdw_1e10(
	uint w, uint dw, uint ws,
	float * _dd_weight,
	float * score_xy, float * score_x, float * score_y, float * score,
	uint sets)
{
	uint set = threadIdx.x + blockIdx.x * blockDim.x;
	if (set < sets)
		_dd_weight[dw*sets*ws + set*ws + w] = 1e10 * (score_xy[set] - score_x[set] - score_y[set] + score[set]);
};

//	ddS = F(x+1e-5,y+1e-5)-F(x+1e-5,y)-F(x,y+1e-5)+F(x,y)
void train_dSdwdw_1e10(Train_t * train, uint start_seed) {
	uint ws = train->mdl->weights;
	uint sets = train->sets;
	uint dws = train->dws;

	float * _score_plus_xy_d;
	SAFE_CUDA(hipMalloc((void**)&_score_plus_xy_d, sizeof(float)*sets));

	float * _score_plus_x_d;
	SAFE_CUDA(hipMalloc((void**)&_score_plus_x_d, sizeof(float)*sets));

	float * _score_plus_y_d;
	SAFE_CUDA(hipMalloc((void**)&_score_plus_y_d, sizeof(float)*sets));

	for (uint dw=0; dw < dws; dw++) {
		for (uint w=0; w < ws; w++) {
			//	F(x+,y+)
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				w, ws, sets, train->_weight, 1e5);
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				train->dw[dw], ws, sets, train->_weight, 1e5);
			train_solo_compute_score(train, start_seed);
			SAFE_CUDA(hipMemcpy(_score_plus_xy_d, train->set_score_d, float(float)*sets, hipMemcpyDeviceToDevice));
			
			//	F(x+)
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				train->dw[dw], ws, sets, train->_weight, -1e5);
			train_solo_compute_score(train, start_seed);
			SAFE_CUDA(hipMemcpy(_score_plus_x_d, train->set_score_d, float(float)*sets, hipMemcpyDeviceToDevice));
			
			//	F(y+)
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				w, ws, sets, train->_weight, -1e5);
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				train->dw[dw], ws, sets, train->_weight, 1e5);
			train_solo_compute_score(train, start_seed);
			SAFE_CUDA(hipMemcpy(_score_plus_y_d, train->set_score_d, float(float)*sets, hipMemcpyDeviceToDevice));

			//	F(x,y)
			_add_x_1e5<<<dim3(KERN_DIV(sets, 32)),dim3(32)>>>(
				train->dw[dw], ws, sets, train->_weight, -1e5);
			train_solo_compute_score(train, start_seed);
			train->set_score

			//	f(x+)-f(x)
			_dSdwdw_1e10<<<dim3(KERN_DIV(train->sets, 32)),dim3(32)>>>(
				w, dw, train->mdl->weights,
				train->_dd_weight,
				_score_plus_xy_d, _score_plus_x_d, _score_plus_y_d, train->set_score_d,
				train->sets
			);
			CUDA_WAIT_KER();
		}
	}
	SAFE_CUDA(hipFree(_score_plus_xy_d));
	SAFE_CUDA(hipFree(_score_plus_x_d));
	SAFE_CUDA(hipFree(_score_plus_y_d));
};