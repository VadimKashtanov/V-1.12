#include "kernel/head/testpackage.cuh"

void test_inst(FILE * fp) {
	float * compare_array;
	
	Train_t * train = test_package_load_train(fp);

	Mdl_t * mdl = train->mdl;
	Data_t * data = train->data;

	uint total = mdl->total;
	uint lines = data->lines;
	uint locds = mdl->locds;
	uint locd2s = mdl->locd2s;
	uint weights = mdl->weights;

	//	Cpu et Use
	compare_array = load_float_array(total * lines, fp);
	is_123(fp);

	printf("==========================================\n");
	printf("================= Cpu_t ==================\n");
	printf("==========================================\n");

	Cpu_t * cpu = cpu_mk(mdl, data);

	cpu_set_input(cpu);
	cpu_forward(cpu);
		
	//	Compare
	cpu_compare_vars(cpu, compare_array);
	if (test_package_compare_cpu_and_cpu(compare_array, cpu->var, mdl->total*data->lines)) {
		OK("Cpu_t->var passed correctly.")
	} else {
		ERR("Il y a des Erreures avec Cpu_t->var")
	}

	cpu_free(cpu);

	printf("==========================================\n");
	printf("================= Use_t ==================\n");
	printf("==========================================\n");

	Use_t * use = use_mk(mdl, data);

	use_set_input(use);
	use_forward(use);

	//	Compare
	use_compare_vars(use, compare_array);
	if (test_package_compare_cpu_and_gpu(compare_array, use->var_d, mdl->total*data->lines)) {
		OK("Use_t->var_d passed correctly.")
	} else {
		ERR("Il y a des Erreures avec Use_t->var_d")
	}

	use_free(use);

	free(compare_array);

	printf("==========================================\n");
	printf("================= Train_t ==================\n");
	printf("==========================================\n");

	uint sets = train->sets;

	const char * arr_names[11] = {
		"_weight",
		"_var",
		
		"_grad",
		"_locd",
		"_meand",
		
		"_dd_weight",
		"_dd_var",
		"_dd_meand",
		"_dd_grad",
		"_dd_locd",
		"_locd2"
	};

	uint lens[11] = {
		sets * weights,
		lines * sets * total,
		lines * sets * total,
		lines * sets * locds,
		sets * weights,

		train->dws * sets * weights,
		lines * sets * total,
		sets * weights,
		lines * sets * total,
		lines * sets * locds,
		lines * sets * locd2s,
	};

	if (train->calcule_d == 1) {
		train_forward_backward(train, 0);

		for (uint i=0; i < 5; i++) {
			printf("=========================================\n");
			printf("=============== %s ============\n", arr_names[i]);

			compare_array = load_float_array(lens[i], fp);
			is_123(fp);

			train_print_compare_arr(train, arr_names[i], compare_array, 0.001);
			if (train_eq_arr(train, arr_names[i], compare_array, 0.001) == 0) {
				ERR("Pas d'egalité suffisante");
			}
		}

		//	On teste juste 1e5 les dSdw. Histoire de reverifier encore un fois
		float * _meand = (float*)malloc(sizeof(float) * weights * sets);
		SAFE_CUDA(hipMemcpy(_meand, train->_meand, sizeof(float)*weights*sets, hipMemcpyDeviceToHost));

		printf("=========================================\n");
		printf("============ Tester _meand 1e5 ==========\n");
		train_dSdw_1e5(train, 0);
		//
		train_print_compare_arr(train, "_meand", _meand, 0.001);
		printf("     dw 1e5 ----- dS (forward-backward)\n");
		if (train_eq_arr(train, "_meand", compare_array, 0.001) == 0) {
			ERR("Pas d'egalité suffisante");
		}

		free(_meand);
	}

	if (train->calcule_dd == 1) {
		train_backward_of_forward_backward(train, 0);

		for (uint i=0; i < 11; i++) {
			printf("=========================================\n");
			printf("=============== %s ============\n", arr_names[i]);

			compare_array = load_float_array(lens[i], fp);
			is_123(fp);

			train_print_compare_arr(train, arr_names[i], compare_array, 0.001);
			if (train_eq_arr(train, arr_names[i], compare_array, 0.001) == 0) {
				ERR("Pas d'egalité suffisante");
			}

			free(compare_array);
		}


		//	On teste juste 1e10 les d(dSdwi)/dwj. Histoire de reverifier encore un fois
		float * _dd_weight = (float*)malloc(sizeof(float) * weights * sets * train->dws);
		SAFE_CUDA(hipMemcpy(_dd_weight, train->_dd_weight, sizeof(float)*train->dws*weights*sets, hipMemcpyDeviceToHost));

		printf("===========================================\n");
		printf("========= Tester _dd_weight 1e10 ==========\n");
		train_dSdwdw_1e10(train, 0);
		//
		train_print_compare_arr(train, "_dd_weight", _dd_weight, 0.001);
		printf("      dwdw 1e10 ----- ddS (forward-backward ** 2)\n");
		if (train_eq_arr(train, "_dd_weight", _dd_weight, 0.001) == 0) {
			ERR("Pas d'egalité suffisante");
		}
		
		free(_dd_weight);
	}

	train_free(train);
	data_free(data);
	mdl_free(mdl);
}