#include "kernel/head/testpackage.cuh"

float * load_float_array(uint len, FILE * fp) {
	float * ret = (float*)malloc(sizeof(float) * len);
	fread(ret, sizeof(float), len, fp);
	return ret;
};

//=====================================================================

static bool compare_arrays(float * cpu0, float * cpu1, uint count)
{
	for (uint i=0; i < count; i++) {
		if (compare_floats(cpu0[i], cpu1[i], 0.01) != true) {
			return false;
		}
	}
	return true;
};

bool test_package_compare_cpu_and_gpu(float * cpu0, float * gpu_d, uint count)
{
	float * cpu = (float*)malloc(sizeof(float) * count);
	SAFE_CUDA(hipMemcpy(cpu, gpu_d, sizeof(float) * count, hipMemcpyDeviceToHost));
	bool ret = compare_arrays(cpu0, cpu, count);
	free(cpu);
	return ret;
};

bool test_package_compare_cpu_and_cpu(float * cpu0, float * cpu1, uint count)
{
	return compare_arrays(cpu0, cpu1, count);
};

//==========================================================================

Data_t * load_test_data(FILE * fp)
{
	uint batchs, lines, inputs, outputs;

	fread(&batchs, sizeof(uint), 1, fp);
	fread(&lines, sizeof(uint), 1, fp);
	fread(&inputs, sizeof(uint), 1, fp);
	fread(&outputs, sizeof(uint), 1, fp);

	Data_t * ret = data_load(batchs, inputs, outputs, lines);

	data_cudamalloc(ret);

	fread(ret->input, sizeof(float), lines*inputs, fp);
	fread(ret->output, sizeof(float), lines*outputs, fp);

	data_cudamemcpy(ret);

	return ret;
};

Train_t * test_package_load_train(FILE * fp) {
	//	Load Contexte
	Mdl_t * mdl = mdl_fp_load(fp);
	is_123(fp);

	Data_t * data = load_test_data(fp);
	is_123(fp);

	uint dw[mdl->weights];
	for (uint i=0; i < mdl->weights; i++) dw[i] = i;

	uint calcule_d, calcule_dd;

	fread(&calcule_d, sizeof(uint), 1, fp);
	is_123(fp);

	fread(&calcule_dd, sizeof(uint), 1, fp);
	is_123(fp);

	Config_t * score = config_load(fp);
	Config_t * opti = config_load(fp);
	Config_t * gtic = config_load(fp);

	Train_t * train = mk_train(
		mdl, data,
		score, opti, gtic,//config_load(fp), config_load(fp), config_load(fp),
		calcule_d,
		calcule_dd,
		mdl->weights, //dws
		dw
	);
	train_random_weights(train, 0);

	float * compare_array = load_float_array(train->sets * mdl->weights, fp);
	is_123(fp);

	printf("==== On verifie simplement que train est initialisé avec les memes weights en python et c/cuda ====\n");
	train_print_compare_arr(train, "_weight", compare_array, 0.001);
	if (train_eq_arr(train, "_weight", compare_array, 0.001) == 0) {
		ERR("Pas egale");
	}

	free(compare_array);

	return train;
};